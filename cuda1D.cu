#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "Eigen/Dense"
#include <cmath>
#define N 1024
#define SIZE 6
#define SIZE_I 3
#define SIZE_E 3

using namespace Eigen;
using namespace std;
typedef Matrix<float, Dynamic, Dynamic, RowMajor> RowMatrixXi;
typedef Eigen::VectorXd RealVector;





__global__ void interaction(float *Mi, float *Mie, float *Mei, float *Me, float *Ki, float *Kie, float *Kei, float *Ke, float *u_old_i, float *u_old_e, float *v_old_i, float *v_old_e,float *a_old_i,float *a_old_e, float *Fe, float *Fi, float * A){


int tid = threadIdx.x + blockIdx.x * blockDim.x;

float delta_t = 0.05;
float dot = 0;

float a_new_e[3];
float u_new_e[3];
float v_new_e[3];

float a_new_i[3];
float u_new_i[3];
float v_new_i[3];


for (int j = 0;j <3;j++){
a_new_e[j] = 0;
a_new_i[j] = 0;

u_new_e[j] = 0;
u_new_e[j] = 0;

v_new_e[j] = 0;
v_new_e[j] = 0;
}


for (int iter = 0; iter <2; iter ++){

for (int j = 0 ; j <3 ; j ++){
dot += Ke[tid*3 + j]*u_old_e[j] -Kei[tid*3 +j]*u_old_i[j];
}


a_new_e[tid] = Fe[tid] - dot;
v_new_e[tid] = v_old_e[tid] + (delta_t/2 * (a_old_e[tid] + a_new_e[tid]));
u_new_e[tid] = u_old_e[tid] + delta_t*v_new_e[tid] + (delta_t*delta_t/2)*a_new_e[tid];


float mult = 0;

__shared__ float temp[3];

for (int j = 0; j<3 ;j++){
mult +=  Kie[tid*3+j]*u_new_e[j] +Ki[tid*3+j]*(u_old_i[j] + delta_t*v_old_i[j] +
(delta_t*delta_t/4)*a_old_i[j]) ;
}

temp[tid] = Fi[tid] - mult;

__syncthreads();

for (int j =0; j <3; j++){
a_new_i[tid] = A[tid*3+j]*temp[j];
}

u_new_i[tid] = u_old_i[tid] + delta_t*v_old_i[tid] + (delta_t*delta_t/4)*(a_old_i[tid] + a_new_i[tid]);
v_new_i[tid] = v_old_i[tid] + (delta_t/2)*(a_old_i[tid] + a_new_i[tid]);


for ( int i = 0 ; i <3 ; i ++){
        u_old_e[i] = u_new_e[i];
        v_old_e[i] = v_new_e[i];
		a_old_e[i] = a_new_e[i];

		u_old_i[i] = u_new_i[i];
		v_old_i[i] = v_new_i[i];
		a_old_i[i] = a_new_i[i];
		}
		
__syncthreads();

   }
   
   

}


void print_arr(float print_arr[][SIZE], int size_row, int size_column){
	for (int i = 0 ; i <size_row; i++){
		for (int j =0; j<size_column; j++){
			cout<<print_arr[i][j];
		}
			cout<<endl;
		}
	}

int main(){




	MatrixXf M(6,6), K(6,6), 
	Me(3,3), Mei(3,3), Mie(3,3), Mi(3,3), 
	Ke(3,3), Kei(3,3), Kie(3,3), Ki(3,3), A(3,3);

	float delta_t = 0.005;

	Vector3f u_old_e, v_old_e, a_old_e, 
	u_new_e, v_new_e, a_new_e,
	u_old_i, v_old_i, a_old_i, 
	u_new_i, v_new_i, a_new_i, Fe, Fi ;

	float u_old_e_host[SIZE_I];
	float u_old_i_host[SIZE_E];
	float v_old_e_host[SIZE_I];
	float v_old_i_host[SIZE_E];
	float a_old_e_host[SIZE_E];
	float a_old_i_host[SIZE_I];
  



	float u_new_e_host[SIZE_I];
	float u_new_i_host[SIZE_E];
	float v_new_e_host[SIZE_I];
	float v_new_i_host[SIZE_E];
	float a_new_e_host[SIZE_E];
	float a_new_i_host[SIZE_I];





	u_old_e << 0, 0, 0;
	u_old_i << 0, 0, 0;
	v_old_i << 0, 0, 0;
	v_old_e << 0, 0, 0;
	a_old_e << 0, 0, 0;
	a_old_i << 0, 0, 0;


	Map<RowMatrixXi>(&u_old_e_host[0], 1, SIZE_E) = u_old_e;
	Map<RowMatrixXi>(&u_old_i_host[0], 1, SIZE_I) = u_old_i;
	Map<RowMatrixXi>(&v_old_e_host[0], 1, SIZE_E) = v_old_e;
	Map<RowMatrixXi>(&v_old_i_host[0], 1, SIZE_I) = v_old_i;
	Map<RowMatrixXi>(&a_old_e_host[0], 1, SIZE_E) = a_old_e;
	Map<RowMatrixXi>(&a_old_i_host[0], 1, SIZE_I) = a_old_i;

	// Map<RowMatrixXi>(&u_new_e_host[0], 1, SIZE_E) = u_new_e;
	// Map<RowMatrixXi>(&u_new_i_host[0], 1, SIZE_I) = u_new_i;
	// Map<RowMatrixXi>(&v_new_e_host[0], 1, SIZE_E) = v_new_e;
	// Map<RowMatrixXi>(&v_new_i_host[0], 1, SIZE_I) = v_new_i;
	// Map<RowMatrixXi>(&a_new_e_host[0], 1, SIZE_E) = a_new_e;
	// Map<RowMatrixXi>(&a_new_i_host[0], 1, SIZE_I) = a_new_i;





	float Fe_host[SIZE_E];
	float Fi_host[SIZE_I];



	Fe << 0, 0.1, 0;
	Fi << 0, 0.2, 0;

	Map<RowMatrixXi>(&Fe_host[0], SIZE_E ,1) = Fe;
	Map<RowMatrixXi>(&Fi_host[0], SIZE_I,1) = Fi;

	//for (int i = 0; i<3 ;i++){
		//cout<<Fe_host[i]<<endl;
	//}



		
		
	M << 1, 0 , 0 , 0 , 0 , 0,
		 0, 1 , 0 , 0 , 0 , 0,
		 0, 0 , 1 , 0 , 0 , 0,
		 0, 0 , 0 , 1 , 0 , 0,
		 0, 0 , 0 , 0 , 1 , 0,
		 0, 0 , 0 , 0 , 0 , 1;

	K << 200, -100, 0, 0, 0, 0,
		 -100, 200, -100, 0, 0, 0,
		 0, -100, 101, -1, 0, 0,
		 0, 0, -1, 2, -1, 0,
		 0, 0, 0, -1, 2, -1,
		 0, 0, 0, 0, -1, 2;

	//float M_host[SIZE][SIZE];
    //float K_host[SIZE][SIZE];

    float Mi_host[9];
    float Mie_host[9];
    float Mei_host[9];
    float Me_host[9];

    float Ki_host[9];
    float Kie_host[9];
    float Kei_host[9];
    float Ke_host[9];
    
    float A_host[9];
	
	Mi = M.block(0,0,3,3);
	Mie = M.block(3,0,3,3);
	Mei = M.block(0,3,3,3);
	Me = M.block(3,3,3,3);
	Ki = K.block(0,0,3,3);
	Kie = K.block(3,0,3,3);
	Kei = K.block(0,3,3,3);
	Ke = K.block(3,3,3,3);
	A = Mi + Ki*(delta_t*delta_t/4);
	A = A.inverse();
	
	Mi.resize(1,9);
	Mie.resize(1,9);
	Mei.resize(1,9);
	Me.resize(1,9);
	
	Ki.resize(1,9);
	Kie.resize(1,9);
	Kei.resize(1,9);
	Ke.resize(1,9);



	 
	//cout << A<<endl;
	A.resize(1,9);

	//Map<RowMatrixXi>(&M_host[0], SIZE, SIZE) = M;
	//Map<RowMatrixXi>(&K_host[0], SIZE, SIZE) = K;

    Map<RowMatrixXi>(&Mi_host[0], 9, 1) = Mi;
    Map<RowMatrixXi>(&Mei_host[0], 9, 1) = Mei;
    Map<RowMatrixXi>(&Mie_host[0], 9, 1) = Mie;
    Map<RowMatrixXi>(&Me_host[0], 9, 1) = Me;

    Map<RowMatrixXi>(&Ki_host[0], 9, 1) = Ki;
    Map<RowMatrixXi>(&Kei_host[0], 9, 1) = Kei;
    Map<RowMatrixXi>(&Kie_host[0], 9, 1) = Kie;
    Map<RowMatrixXi>(&Ke_host[0], 9,1) = Ke;
    
    Map<RowMatrixXi>(&A_host[0], 9, 1) = A;
    
//    for (int i = 0; i <9; i++){
//  cout<<A_host[i]<<endl;}



    float *u_old_i_device;
    float *u_old_e_device;
    float *v_old_i_device;
    float *v_old_e_device;
    float *a_old_i_device;
    float *a_old_e_device;
    
    float *A_device;
    
    float *Mi_device;
    float *Mie_device;
    float *Mei_device;
    float *Me_device;
    
    float *Ki_device;
    float *Kie_device;
    float *Kei_device;
    float *Ke_device;
    
    float *Fe_device;
    float *Fi_device; 
    
    //hipMalloc((void**)&Fe_device, 3*sizeof(float));
    //hipMalloc((void**)&Fe_trial, 3*sizeof(float));
        
    hipMalloc((void**)&u_old_i_device, 3*sizeof(float));
    hipMalloc((void**)&u_old_e_device, 3*sizeof(float));
    hipMalloc((void**)&v_old_i_device, 3*sizeof(float));
    hipMalloc((void**)&v_old_e_device, 3*sizeof(float));
    hipMalloc((void**)&a_old_i_device, 3*sizeof(float));
    hipMalloc((void**)&a_old_e_device, 3*sizeof(float));
    
    
    hipMalloc((void**)&Mi_device, 9*sizeof(float));
    hipMalloc((void**)&Mie_device, 9*sizeof(float));
    hipMalloc((void**)&Mei_device, 9*sizeof(float));
    hipMalloc((void**)&Me_device, 9*sizeof(float));
    
    
    hipMalloc((void**)&Ki_device, 9*sizeof(float));
    hipMalloc((void**)&Kie_device, 9*sizeof(float));
    hipMalloc((void**)&Kei_device, 9*sizeof(float));
    hipMalloc((void**)&Ke_device, 9*sizeof(float));
    
    hipMalloc((void**)&A_device, 9*sizeof(float));
    
    hipMalloc((void**)&Fe_device, 3*sizeof(float));
    hipMalloc((void**)&Fi_device, 3*sizeof(float));
          
               
   
    hipMemcpy(u_old_i_device, &u_old_i_host[0],3*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(u_old_e_device, &u_old_e_host[0],3*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(v_old_i_device, &v_old_i_host[0],3*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(v_old_e_device, &v_old_e_host[0],3*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(a_old_i_device, &a_old_i_host[0],3*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(a_old_e_device, &a_old_e_host[0],3*sizeof(float),hipMemcpyHostToDevice);
    
    
    
    hipMemcpy(Ki_device, &Ki_host[0],9*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Kie_device, &Kie_host[0],9*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Kei_device, &Kei_host[0],9*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Ke_device, &Ke_host[0],9*sizeof(float),hipMemcpyHostToDevice);
     
        
    hipMemcpy(Mi_device, &Mi_host[0],9*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Mie_device, &Mie_host[0],9*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Mei_device, &Mei_host[0],9*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Me_device, &Me_host[0],9*sizeof(float),hipMemcpyHostToDevice);
    
    hipMemcpy(A_device,&A_host[0], 9*sizeof(float),hipMemcpyHostToDevice);
    

    hipMemcpy(Fe_device,&Fe_host[0], 3*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Fi_device,&Fi_host[0], 3*sizeof(float),hipMemcpyHostToDevice);
    
    
    //interaction<<<1,1>>>(Fe_device,Fe_trial);
    //float trial[3];
    
    
    
    //hipMemcpy(trial,Fe_trial, 3*sizeof(float),hipMemcpyDeviceToHost);
    
    //cout<<trial[0]<<endl;
    
    //cout<<trial[1]<<endl;
    //cout<<trial[2]<<endl;
    
    interaction <<<1,3>>>(Mi_device, Mie_device, Mei_device, Me_device, Ki_device, Kie_device, Kei_device, Ke_device, u_old_i_device, u_old_e_device, v_old_i_device, v_old_e_device,
    a_old_i_device,a_old_e_device,Fe_device, Fi_device, A_device);
    
    hipMemcpy(&a_new_e_host[0], a_old_e_device, 3*sizeof(float), hipMemcpyDeviceToHost);
    
    for ( int k = 0 ; k < 3; k++)
    {cout <<a_new_e_host[k]<<endl;}



	

/*	for (int i = 0; i < 2; i++) {
		a_new_e = (Fe - Ke*u_old_e - Kei*u_old_i);
		v_new_e = v_old_e + (delta_t/2) * ( a_old_e + a_new_e);
		u_new_e = u_old_e + delta_t*v_new_e + (delta_t*delta_t/2)*a_new_e;

		a_new_i = A*(Fi - Kie*u_new_e - Ki*(u_old_i + delta_t*v_old_i + (delta_t*delta_t/4)*a_old_i));
		u_new_i = u_old_i + delta_t*v_old_i + (delta_t*delta_t/4)*(a_old_i + a_new_i);
		v_new_i = v_old_i + (delta_t/2)*(a_old_i+a_new_i);

		u_old_e = u_new_e;
		v_old_e = v_new_e;
		a_old_e = a_new_e;

		u_old_i = u_new_i;
		v_old_i = v_new_i;
		a_old_i = a_new_i;

	}
	*/
	//cout << a_new_e << endl;

	


}
